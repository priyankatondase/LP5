#include "hip/hip_runtime.h"
%%cuda --name omp_cuda.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

#define row1 2 /* Number of rows of first matrix */
#define col1 3 /* Number of columns of first matrix */
#define row2 3 /* Number of rows of second matrix */
#define col2 2 /* Number of columns of second matrix */

__global__ void matproduct(int *l, int *m, int *n)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int k;

    n[col2 * y + x] = 0;
    for (k = 0; k < col1; k++)
    {
        n[col2 * y + x] = n[col2 * y + x] + l[col1 * y + k] * m[col2 * k + x];
    }
}

__global__ void arradd(int *x, int *y, int *z)
{
    int id = blockIdx.x;
    z[id] = x[id] + y[id];
}

int main()
{
    int a[row1][col1];
    int b[row2][col2];
    int c[row1][col2];
    int *d, *e, *f;
    int i, j;

    printf("\n Enter elements of first matrix of size 2*3\n");
    for (i = 0; i < row1; i++)
    {
        for (j = 0; j < col1; j++)
        {
            scanf("%d", &a[i][j]);
        }
    }
    printf("\n Enter elements of second matrix of size 3*2\n");
    for (i = 0; i < row2; i++)
    {
        for (j = 0; j < col2; j++)
        {
            scanf("%d", &b[i][j]);
        }
    }

    hipMalloc((void **)&d, row1 * col1 * sizeof(int));
    hipMalloc((void **)&e, row2 * col2 * sizeof(int));
    hipMalloc((void **)&f, row1 * col2 * sizeof(int));

    hipMemcpy(d, a, row1 * col1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(e, b, row2 * col2 * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(col2, row1);

    matproduct<<<grid, 1>>>(d, e, f);

    hipMemcpy(c, f, row1 * col2 * sizeof(int), hipMemcpyDeviceToHost);
    printf("\nProduct of two matrices:\n ");
    for (i = 0; i < row1; i++)
    {
        for (j = 0; j < col2; j++)
        {
            printf("%d\t", c[i][j]);
        }
        printf("\n");
    }

    int arr[6];
    int arr2[6];
    int arr3[6];

    printf("\n Enter six elements of first array\n");
    for (i = 0; i < 6; i++)
    {
        scanf("%d", &arr[i]);
    }
    printf("\n Enter six elements of second array\n");
    for (i = 0; i < 6; i++)
    {
        scanf("%d", &arr2[i]);
    }

    hipMemcpy(d, arr, 6 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(e, arr2, 6 * sizeof(int), hipMemcpyHostToDevice);

    arradd<<<6, 1>>>(d, e, f);

    hipMemcpy(arr3, f, 6 * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nSum of two arrays:\n ");
    for (i = 0; i < 6; i++)
    {
        printf("%d\t", arr3[i]);
    }

    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}
